
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <functional>
#include <fstream>
#include <vector>
#include <sstream>
#include <random>
#include <chrono>

// Program includes
// #include "ga_gpu.h"
// #include "common.h"

using namespace std;

struct City
{
//Store location - city
	int x, y;
};

typedef struct World
{
	// 2D world for the TSP
	int num_cities;    
	City* cities;      
	float fitness;     // The current fitness
	float fit_prob;    // The fitness probability

	inline __host__ void calc_fitness()
	{
		float distance = 0.0;
		for (int i=0; i<num_cities-1; i++)
			distance += (cities[i].x - cities[i + 1].x) * (cities[i].x -      \
				cities[i +1 ].x) + (cities[i].y - cities[i + 1].y)     *      \
				(cities[i].y - cities[i + 1].y);
		fitness = 1.0 / distance;
	}

	inline __host__ float calc_distance()
	{
		float distance = 0.0;
		for (int i=0; i<num_cities-1; i++)
			distance += (float)sqrt((float)((cities[i].x - cities[i + 1].x) * \
				(cities[i].x - cities[i + 1].x) + (cities[i].y              - \
				cities[i +1 ].y) * (cities[i].y - cities[i + 1].y)));
		return distance;
	}
} World;

bool checkForError(hipError_t error)
{
	if (error != hipSuccess)
	{
		cout << hipGetErrorString(error) << endl;
		return true;
	}
	else
	{
		return false;
	}
}

bool checkForKernelError(const char *err_msg)
{
	hipError_t status = hipGetLastError();
	if (status != hipSuccess)
	{
		cout << err_msg << hipGetErrorString(status) << endl;
		return true;
	}
	else
	{
		return false;
	}
}

//Read data
vector<pair<int, int>> get_coor(string filename){
    ifstream dataFile(filename, ios::in);
    if(!dataFile.is_open()){
        cout<<"file does not exit!";
    }
    vector<pair<int, int>> coor;
    string data_line;
    while(getline(dataFile, data_line)){
        if(data_line == "EOF")   break;
        istringstream ss(data_line);
        string data;
        ss >> data;
        if(isdigit(data[0])){
            string x, y;
            ss >> x;
            ss >> y;
            coor.push_back(make_pair(stoi(x), stoi(y)));
        }
    }
    return coor;
}

void init_world(World* world, int num_cities)
{
	world->num_cities = num_cities;
	world->fitness    = (float)0.0;
	world->fit_prob   = (float)0.0;
	world->cities     = new City[num_cities * sizeof(City)];
}

void clone_city(City* src, City* dst, int num_cities)
{
memcpy(dst, src, num_cities * sizeof(City));
}

void clone_world(World* src, World* dst)
{
	dst->num_cities = src->num_cities;
	dst->fitness    = src->fitness;
	dst->fit_prob   = src->fit_prob;
	clone_city(src->cities, dst->cities, src->num_cities);
}

void free_world(World* world)
{
	delete[] world->cities;
	delete[] world;
}

//Functions for both CPU and GPU
void make_world(World* world, string filename, int seed)
{
	// Random number generation
	mt19937::result_type rseed = seed;
	auto rgen = bind(uniform_real_distribution<>(0, 1), mt19937(rseed));
	
	// Create a set to deal with uniqueness
	vector<pair<int, int>> coordinates = get_coor(filename);
	vector<pair<int, int>>::iterator it;
	
	init_world(world,coordinates.size());
	// Add those cities to the world
	{
		int i = 0;
		for (it=coordinates.begin(); it!=coordinates.end(); it++)
		{
			world->cities[i].x = (*it).first;
			world->cities[i].y = (*it).second;
			i++;
		}
	}
}

bool g_soft_clone_world(World* d_world, World* h_world)
{
	// Error checking
	bool error;
	
	// error = checkForError(cudaMemcpy(&d_world->width, &h_world->width,        \
	// 	sizeof(int), cudaMemcpyHostToDevice));
	// if (error)
	// return true;
	// error = checkForError(cudaMemcpy(&d_world->height, &h_world->height,      \
	// 	sizeof(int), cudaMemcpyHostToDevice));
	// if (error)
	// return true;
	error = checkForError(hipMemcpy(&d_world->num_cities,                    \
		&h_world->num_cities, sizeof(int), hipMemcpyHostToDevice));
	if (error)
	return true;

	return false;
}

bool g_init_world(World* d_world, World* h_world)
{
	// Error checking
	bool error;
	
	// Soft clone world
	error = g_soft_clone_world(d_world, h_world);
	if (error)
		return true;
	
	// Allocate space for cities on device
	City *d_city;
	error = checkForError(hipMalloc((void**)&d_city, h_world->num_cities * sizeof(City)));
	if (error)
	return true;
	
	// Update pointer on device
	error = checkForError(hipMemcpy(&d_world->cities, &d_city, sizeof(City*), hipMemcpyHostToDevice));
	if (error)
	return true;
	
	return false;
}

void print_status(World* generation_leader, World* best_leader, int generation)
{
	cout << "Generation " << generation << ":" << endl;
	cout << "  Current Leader's Fitness: "  << generation_leader->fitness << endl;
	cout << "  Best Leader's Fitness: "  << best_leader->fitness << endl;
}

void storeBestLeader(World* bestleader, string path){
	ofstream resFile(path, ios::out);
	for(int i = 0;i<bestleader->num_cities;i++){
		resFile<<bestleader->cities[i].x<<" "<<bestleader->cities[i].y<<endl;
	}
	resFile.close();
}

__device__ int getGlobalIdx_2D_1D()
{
	int blockId  = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x; 
	return threadId;
}

__device__ void crossover(World* old_pop, World* new_pop, int* sel_ix,  int* cross_loc, int tid)
{

	// Copy elements from first parent up through crossover point
	memcpy(new_pop[tid].cities, old_pop[sel_ix[2*tid]].cities,(cross_loc[tid] + 1) * sizeof(City));

	// Add remaining elements from second parent to child, in order
	int remaining = old_pop[tid].num_cities - cross_loc[tid] - 1;
	int count     = 0;
	for (int i=0; i<old_pop[tid].num_cities; i++)
	{
		bool in_child = false;
		for (int j=0; j<=cross_loc[tid]; j++)     
		{
			// If the city is in the child, exit
			if ((new_pop[tid].cities[j].x == old_pop[sel_ix[2 * tid + 1]].cities[i].x) &               
				(new_pop[tid].cities[j].y == old_pop[sel_ix[2 * tid + 1]].cities[i].y))
			{
				in_child = true;
				break;
			}
		}
		if (!in_child)
		{
			count++;
			memcpy(&new_pop[tid].cities[cross_loc[tid] + count],&old_pop[sel_ix[2 * tid + 1]].cities[i], sizeof(City));
		}
	
		// Stop once all of the cities have been added
		if (count == remaining) break;
	}
}

__device__ void mutate(World* new_pop, int* mutate_loc, int tid)
{
	// Swap the elements
	City temp = *(new_pop[tid].cities + mutate_loc[2*tid]);
	*(new_pop[tid].cities + mutate_loc[2*tid])   = *(new_pop[tid].cities + mutate_loc[2*tid+1]);
	*(new_pop[tid].cities + mutate_loc[2*tid+1]) = temp;
}

__global__ void fitness_kernel(World* pop, int pop_size)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();
	
	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		float distance = (float)0.0; // Total "normalized" "distance"
		// Calculate fitnesses using the fitness formula that will be explained in the reports
		int N = pop[tid].num_cities;
		for (int i=0; i<pop[tid].num_cities-1; i++){
			distance += (pop[tid].cities[i].x - pop[tid].cities[(i + 1)%N].x) * (pop[tid].cities[i].x - pop[tid].cities[(i + 1)%N].x)         
				+ (pop[tid].cities[i].y - pop[tid].cities[(i + 1)%N].y)*(pop[tid].cities[i].y - pop[tid].cities[(i + 1)%N].y);
		}
		pop[tid].fitness = 3950.0 * 1969 / distance;
	}
}

__global__ void fit_sum_kernel(World* pop, int pop_size, float* fit_sum)
{
	int tid = getGlobalIdx_2D_1D();
	
	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		// Sum of all fitness
		float sum = (float)0.0;
		
		// Calculate the partial sum
		for (int i=0; i<=tid; i++)
			sum += pop[i].fitness;
		// printf("sum:%.10f\n", sum);
		pop[tid].fit_prob = sum;

		if (tid == (pop_size - 1))	*fit_sum = sum;
	}
}

__device__ float max_fitsum_kernel(int pop_size, float* fit_sum){
	float max = fit_sum[0];
	for(int i = 0;i<pop_size;i++){
		if(fit_sum[i]>max)	max = fit_sum[i];
	}
	return max;
}

__device__ float min_fitsum_kernel(int pop_size, float* fit_sum){
	float min = fit_sum[0];
	for(int i = 0;i<pop_size;i++){
		if(fit_sum[i]<min)	min = fit_sum[i];
	}
	return min;
}

__global__ void fit_prob_kernel(World* pop, int pop_size, float* fit_sum)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();
	// float fit_sum_max = max_fitsum_kernel(pop_size, fit_sum);
	// float fit_sum_min = min_fitsum_kernel(pop_size, fit_sum);
	float factor = max_fitsum_kernel(pop_size, fit_sum) - min_fitsum_kernel(pop_size, fit_sum);
	// printf("factor:%.10f\n", factor);
	// Evaluate if the thread is valid
	if (tid < pop_size)
		pop[tid].fit_prob /= *fit_sum;
		// printf("fit_sum:%.10f, prob:%.10f\n", *fit_sum, pop[tid].fit_prob);
}

__global__ void max_fit_kernel(World* pop, int pop_size, World* gen_leader)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		if (tid == 0)
		{
			float max = (float)0.0;
			int ix  = 0;
			for (int i=1; i<pop_size; i++)
			{
				if (pop[i].fitness > max)
				{
					max = pop[i].fitness;
					ix  = i;
				}
			}
			gen_leader->cities  = pop[ix].cities;
			gen_leader->fitness = max;
		}
		else if (tid == 1)
		{
			gen_leader->num_cities = pop[0].num_cities;
		}
	}
}

__global__ void selection_kernel(World* pop, int pop_size, float* rand_nums,  \
	int* sel_ix)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < (2 * pop_size))
	{
		// Select the parents
		for (int j=0; j<pop_size; j++)
		{
			if (rand_nums[tid] <= pop[j].fit_prob)
			{
				sel_ix[tid] = j;
				break;
			}
		}
	}
}

__global__ void child_kernel(World* old_pop, World* new_pop, int pop_size,    \
	int* sel_ix, float prob_crossover, float* prob_cross, int* cross_loc,     \
	float prob_mutation, float* prob_mutate, int* mutate_loc)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		// Determine how many children are born
		if (prob_cross[tid] <= prob_crossover)
		{
			crossover(old_pop, new_pop, sel_ix, cross_loc, tid);
			if(prob_mutate[tid] <= prob_mutation)
				mutate(new_pop, mutate_loc, tid);
			
		}
		else // Select the first parent
		{
			// Add child to new population
			memcpy(new_pop[tid].cities, old_pop[sel_ix[2*tid]].cities, old_pop[tid].num_cities * sizeof(City));
			if(prob_mutate[tid] <= prob_mutation)
				mutate(new_pop, mutate_loc, tid);
		}
	}
}

bool g_initialize(World* world, World* pop, int pop_size, int seed)
{
	bool error;
	World h_world;
	h_world.cities = new City[world->num_cities * sizeof(City)];
	srand(seed);
	for (int i=0; i<pop_size; i++)
	{
		clone_world(world, &h_world);
		// Randomly adjust the path between cities
		random_shuffle(&h_world.cities[0], &h_world.cities[world->num_cities]);
		// Copy world to device
		error = g_soft_clone_world(&pop[i], &h_world);
		if (error)
		return true;

		// Allocate space for cities on device
		City *d_city;
		error = checkForError(hipMalloc((void**)&d_city, world->num_cities * sizeof(City)));
		if (error)
		return true;

		// Copy cities to device
		error = checkForError(hipMemcpy(d_city, h_world.cities,world->num_cities * sizeof(City), hipMemcpyHostToDevice));
		if (error)
		return true;

		// Update pointer on device
		error = checkForError(hipMemcpy(&pop[i].cities, &d_city, sizeof(City*), hipMemcpyHostToDevice));
		if (error)
		return true;
	}

	// Success
	delete[] h_world.cities; return false;
}

bool g_evaluate(World *pop, int pop_size, dim3 Block, dim3 Grid)
{
	bool error;
	
	// Allocate fitness sum on the GPU
	float *fit_sum_d;
	error = checkForError(hipMalloc((void**)&fit_sum_d, sizeof(float)));
	if (error)
	return true;
	
	// Calculate the fitnesses
	fitness_kernel <<< Grid, Block >>> (pop, pop_size);
	hipDeviceSynchronize();
	if (checkForKernelError("fitness_kernel is failing "))
		return true;
	// Calculate the total sum and compute the partial probabilities
	fit_sum_kernel <<< Grid, Block >>> (pop, pop_size, fit_sum_d);
	hipDeviceSynchronize();
	if (checkForKernelError("fit_sum_kernel failing "))
		return true;

	// Compute the full probabilities
	fit_prob_kernel <<< Grid, Block >>> (pop, pop_size, fit_sum_d);
	hipDeviceSynchronize();
	if (checkForKernelError("fit_prob_kernel failing "))
		return true;

	hipFree(fit_sum_d); 
	return false;
}

int g_select_leader(World* pop, int pop_size, World* generation_leader,
	World* best_leader, dim3 Block, dim3 Grid)
{
	// Error handling
	bool error;

	// Initialize world for device generation leader
	World *gen_leader_d;
	error = checkForError(hipMalloc((void**)&gen_leader_d, sizeof(World)));
	if (error)
	return -1;
	// Purposely don't allocate space for the cities, simply use a pointer.
	// Since we only need to copy data to cpu 
	error = g_soft_clone_world(gen_leader_d, generation_leader);
	if (error)
	return -1;

	// Calculate the max fitness
	max_fit_kernel <<< Grid, Block >>> (pop, pop_size, gen_leader_d);
	hipDeviceSynchronize();
	if (checkForKernelError("*** Max fitness kernel failed: "))
	return -1;

	// Copy results from device
	City *h_ptr = generation_leader->cities;
	City *d_ptr;
	error = checkForError(hipMemcpy(generation_leader, gen_leader_d, sizeof(World), hipMemcpyDeviceToHost));
	if (error)
	{
		cout<<"memcpy generation leader error!"<<endl;
		return -1;
	}
	d_ptr = generation_leader->cities;
	generation_leader->cities = h_ptr;
	error = checkForError(hipMemcpy(generation_leader->cities,  d_ptr, generation_leader->num_cities * sizeof(City), hipMemcpyDeviceToHost));
	if (error){
		cout<<"memcpy cities error!"<<endl;
		return -1;
	}

	// Update best leader
	if (generation_leader->fitness > best_leader->fitness)
	{
		clone_world(generation_leader, best_leader);
		hipFree(gen_leader_d); return 1;
	}

	// Success
	hipFree(gen_leader_d);	return 0;
}

bool g_execute(float prob_mutation, float prob_crossover, int pop_size,
	int max_gen, World* world, int seed)
{
	hipSetDevice(2);
	// Error checking variables
	bool error;
	
	// Random number generation
	mt19937::result_type rseed = seed;
	auto rgen = bind(uniform_real_distribution<>(0, 1), mt19937(rseed));
	
	// Tile and grid variables
	int tile_size  = 1024;
	int grid_size  = (int)ceil((float)pop_size / tile_size);
	int grid_size2 = (int)ceil((float)(2 * pop_size) / tile_size);
	dim3 Block(tile_size);
	dim3 Grid(grid_size, grid_size);
	dim3 Grid2(grid_size2, grid_size2);
	
	int pop_bytes  = pop_size * sizeof(World);
	World *old_pop_d, *new_pop_d;

	// Random numbers
	float *prob_select = new float[2 * pop_size * sizeof(float)];
	float *prob_cross  = new float[pop_size * sizeof(float)];
	float *prob_mutate = new float[pop_size * sizeof(float)];
	int   *cross_loc   = new int[pop_size * sizeof(int)];
	int   *mutate_loc  = new int[2 * pop_size * sizeof(int)];
	float *prob_select_d, *prob_cross_d, *prob_mutate_d;
	int   *cross_loc_d, *mutate_loc_d;
	
	int   sel;
	int   best_generation    = 0;
	World *best_leader       = new World[sizeof(World)];
	World *generation_leader = new World[sizeof(World)];

	// Other "temporary" parameters
	int *sel_ix_d;

	//Initializations
	init_world(best_leader, world->num_cities);
	init_world(generation_leader, world->num_cities);
	if (checkForError(hipMalloc((void**) &old_pop_d, pop_bytes)))
	return true;
	if (checkForError(hipMalloc((void**) &new_pop_d, pop_bytes)))
	return true;
	if (checkForError(hipMalloc((void**) &prob_select_d, sizeof(float)* 2 * pop_size)))
	return true;
	if (checkForError(hipMalloc((void**) &prob_cross_d, sizeof(float) * pop_size)))
	return true;
	if (checkForError(hipMalloc((void**) &prob_mutate_d, sizeof(float) * pop_size)))
	return true;
	if (checkForError(hipMalloc((void**) &cross_loc_d, sizeof(int) * pop_size)))
	return true;
	if (checkForError(hipMalloc((void**) &mutate_loc_d, sizeof(int) * 2 * pop_size)))
	return true;
	// Other parameters
	if (checkForError(hipMalloc((void**) &sel_ix_d, sizeof(int) * 2 * pop_size)))
	return true;

	///////// GPU Initializations
	// Populations
	error = g_initialize(world, old_pop_d, pop_size, seed);
	if (error)
	return true;
	for (int i=0; i<pop_size; i++)
	{
		error = g_init_world(&new_pop_d[i], world);
		if (error)
		return true;
	}

	// Calculate the fitnesses
	error = g_evaluate(old_pop_d, pop_size, Block, Grid);
	if (error)
	return true;
	
	// Initialize the best leader
	sel = g_select_leader(old_pop_d, pop_size, generation_leader,             \
		best_leader, Block, Grid);	
	if (-1 == sel)
	return true;	
	print_status(generation_leader, best_leader, 0);
		
	// Continue through all generations
	for (int i=0; i<max_gen; i++)
	{

		// Generate all probabilities for each step
		// Note : The order the random numbers are generated must be consistent to
		// ensure the results will match the CPU.
		for (int j=0; j<pop_size; j++)
		{
			prob_select[2*j]     = (float)rgen();
			prob_select[2*j + 1] = (float)rgen();
			prob_cross[j]        = (float)rgen();
			cross_loc[j]         = (int)(rgen() * (world->num_cities - 1));
			prob_mutate[j]       = (float)rgen();
			mutate_loc[2*j]      = (int)(rgen() * (world->num_cities));
			mutate_loc[2*j + 1]  = (int)(rgen() * (world->num_cities));
			while (mutate_loc[2*j + 1] == mutate_loc[2*j])
			{
				mutate_loc[2*j + 1] = (int)(rgen() * world->num_cities);
			}
		}
		
		// Copy random numbers to device
		if (checkForError(hipMemcpy(prob_select_d, prob_select, 2 * pop_size * sizeof(float), hipMemcpyHostToDevice)))
		return true;
		if (checkForError(hipMemcpy(prob_cross_d, prob_cross, pop_size * sizeof(float), hipMemcpyHostToDevice)))
		return true;
		if (checkForError(hipMemcpy(prob_mutate_d, prob_mutate,pop_size * sizeof(float), hipMemcpyHostToDevice)))
		return true;
		if (checkForError(hipMemcpy(cross_loc_d, cross_loc,pop_size * sizeof(int), hipMemcpyHostToDevice)))
		return true;
		if (checkForError(hipMemcpy(mutate_loc_d, mutate_loc, 2 * pop_size * sizeof(int), hipMemcpyHostToDevice)))
		return true;

		// Select the parents
		selection_kernel <<< Grid2, Block >>> (old_pop_d, pop_size, prob_select_d, sel_ix_d);
		hipDeviceSynchronize();
		if (checkForKernelError("slection_kernel is failing "))
		return true;
		
		// Create the children (form the new population entirely on the GPU!)
		child_kernel <<< Grid, Block >>> (old_pop_d, new_pop_d, pop_size, sel_ix_d, prob_crossover, prob_cross_d, cross_loc_d, prob_mutation, prob_mutate_d, mutate_loc_d);
		hipDeviceSynchronize();
		if (checkForKernelError("child_kernel is failing"))
		return true;
		
		// Calculate the fitnesses on the new population
		error = g_evaluate(new_pop_d, pop_size, Block, Grid);
		if (error)
		return true;	

		// Swap the populations
		World* temp_d = old_pop_d;
		old_pop_d     = new_pop_d;
		new_pop_d     = temp_d;

		// Select the new leaders
		sel = g_select_leader(old_pop_d, pop_size, generation_leader,best_leader, Block, Grid);
		if (-1 == sel)
		return true;
		else if (1 == sel) best_generation = i + 1;
		print_status(generation_leader, best_leader, i + 1);
	} 
	
	cout << endl << "Best generation found at " << best_generation << " generations" << endl;

	storeBestLeader(best_leader, "/home/TSP/res/bestleader.txt");

	//freeing memory
	delete[] prob_select; delete[] prob_cross; delete[] prob_mutate;
	delete[] cross_loc; delete[] mutate_loc; free_world(best_leader);
	free_world(generation_leader); hipFree(old_pop_d); hipFree(cross_loc_d); 
	hipFree(new_pop_d); hipFree(prob_select_d); hipFree(prob_cross_d);
	hipFree(prob_mutate_d); hipFree(mutate_loc_d); hipFree(sel_ix_d);	
	return false;
}


int main()
{
	// GA parameters
	float prob_mutation  = (float)0.15; // The probability of a mutation
	float prob_crossover = (float)0.8;  // The probability of a crossover
	int world_seed       = 12438955;    // Seed for initial city selection
	int ga_seed          = 87651111;    // Seed for all other random numbers
	
	// The test cases
	int iterations          = 1;  // Number of full runs
	const int num_cases     = 1; // How many trials to test
	int cases[num_cases][2] =     // pop_size, max_gen
	{
		{100, 10000}
	};
	for (int i=0; i<num_cases; i++)
	{
		int pop_size   = cases[i][0];
		int max_gen    = cases[i][1];
		string filename = "/home/TSP/TSPlib/kroA100.tsp";
		World* world = new World[sizeof(World)];
		make_world(world, filename, world_seed);
		cout << "GPU Version - START" << endl;
		auto start = std::chrono::steady_clock::now();
		for (int j=0; j<iterations; j++)
		{
			hipDeviceReset(); 
			if(g_execute(prob_mutation, prob_crossover, pop_size, max_gen,world, ga_seed))
				{
					cout<<"GPU Related error - Could be an issue if GPU is being used by others"<<endl
						<<"Please try running again when there is memory free + GPU is free"	<<endl;
				}

		}
		auto end = std::chrono::steady_clock::now();
		auto time_used1 = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
		cout<<"use"<<time_used1.count()<<"s"<<endl;
		hipDeviceReset();

		cout << "GPU Version - END" << endl;

		free_world(world);
	}
	
	return 0;
}
