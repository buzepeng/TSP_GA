#include "hip/hip_runtime.h"
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include <cstdio>
#include <sstream>
#include <fstream>
#include <iostream>
#include <string.h>

#include "TSP_GA.cuh"
#include "utils.h"
#include "utils.cuh"

void TSP_GA::InitCityAndPop(std::string filename, int s, float vx, float vy, float tx, float ty){
    //read city coordinates
    std::ifstream fp(filename, std::ios::in);
    std::string line;
    thrust::host_vector<int2>   h_city_coors;
    if(!fp.is_open()){
        std::cout<<"Error: opening file fail"<<std::endl;
        std::exit(1);
    }
    int cnt = 0;
    while(std::getline(fp, line)){
        std::istringstream sin(line);
        std::string temp;

        sin>>temp;
        if(!isStringNumber(temp)){
            sin>>temp;
            if(isStringNumber(temp)){
                N = std::stoi(temp);
                h_city_coors.resize(N);
            }
            continue;
        }
        
        sin>>h_city_coors[cnt].x>>h_city_coors[cnt].y;
        cnt++;
    }
    fp.close();

    //compute city distance
    thrust::host_vector<float> city_distance(N*N), coor_x_tmp1(N*N), coor_x_tmp2(N*N), coor_y_tmp1(N*N), coor_y_tmp2(N*N), lx(N*N), ly(N*N);
    for(int i = 0;i<N;i++){
        for(int j = 0;j<N;j++){
            coor_x_tmp1[i*N+j] = h_city_coors[i].x;
            coor_x_tmp2[j*N+i] = h_city_coors[i].x;
            coor_y_tmp1[i*N+j] = h_city_coors[i].y;
            coor_y_tmp2[j*N+i] = h_city_coors[i].y;
        }
    }
    thrust::transform(thrust::host, coor_x_tmp1.begin(), coor_x_tmp1.end(), coor_x_tmp2.begin(), lx.begin(), [=]__host__(float tmp1, float tmp2){
        return (double)(abs(tmp1-tmp2)/vx+tx);
    });
    thrust::transform(thrust::host, coor_y_tmp1.begin(), coor_y_tmp1.end(), coor_y_tmp2.begin(), ly.begin(), [=]__host__(float tmp1, float tmp2){
        return (double)(abs(tmp1-tmp2)/vy+ty);
    });
    thrust::transform(thrust::host, lx.begin(), lx.end(), ly.begin(), city_distance.begin(), [=]__host__(float x, float y){
        return max(x, y);
    });
    city_dis = city_distance;

    //generate population
    thrust::host_vector<int> h_pop(s*N, 0);
    for(int i = 0;i<s;i++){
        int sb = rand() % N;
        std::vector<bool> visited(N, false);
        visited[sb] = true;
        int *path = h_pop.data()+i*N;
        path[0] = sb;
        for(int j = 0;j<N-1;j++){
            thrust::host_vector<float> q(city_distance.begin()+sb*N, city_distance.begin()+(sb+1)*N);
            for(int k = 0;k<N;k++)  if(visited[k])  q[k] = FLT_MAX;
            int location = thrust::min_element(thrust::host, q.begin(), q.end())-q.begin();
            path[j+1] = location;
            visited[location] = true;
            sb = location;
        }
    }
    
    new_pop = h_pop;
    old_pop.resize(s*N);
    old_fit.resize(s);
    new_fit.resize(s);
    distance.resize(s);
    pop_min.resize(N);
}

thrust::host_vector<int> TSP_GA::find_shortest(int s, float c, float pc, float pm, int times){
    thrust::host_vector<float> h_prob_a(s), h_prob_b(s);
    thrust::host_vector<int> h_cross_pos(2*s), h_mutation_pos(2*s);
    thrust::device_vector<float> pc1(int(s/2)),pm1(s), d_prob_a(s), d_prob_b(s);
    thrust::device_vector<int> old_ind(s), new_ind(s), d_cross_pos(2*s), d_mutation_pos(2*s);
    thrust::device_vector<int> ind(s);
    thrust::sequence(thrust::device, ind.begin(), ind.end());
    int *old_pop_ptr = thrust::raw_pointer_cast(old_pop.data());
    int *new_pop_ptr = thrust::raw_pointer_cast(new_pop.data());
    int *old_ind_ptr = thrust::raw_pointer_cast(old_ind.data());
    int *new_ind_ptr = thrust::raw_pointer_cast(new_ind.data());
    int *pop_min_ptr = thrust::raw_pointer_cast(pop_min.data());
    int *cross_pos_ptr = thrust::raw_pointer_cast(d_cross_pos.data());
    int *mutation_pos_ptr = thrust::raw_pointer_cast(d_mutation_pos.data());
    float *pc1_ptr = thrust::raw_pointer_cast(pc1.data());
    float *pm1_ptr = thrust::raw_pointer_cast(pm1.data());
    float *old_fit_ptr = thrust::raw_pointer_cast(old_fit.data());
    float *new_fit_ptr = thrust::raw_pointer_cast(new_fit.data());
    float *city_dis_ptr = thrust::raw_pointer_cast(city_dis.data());
    float *prob_a_ptr = thrust::raw_pointer_cast(d_prob_a.data());
    float *prob_b_ptr = thrust::raw_pointer_cast(d_prob_b.data());
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(0), new_fit.begin(), old_fit.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(s), new_fit.end(), old_fit.end()));

    int next_pow2N = getCeilPowerOfTwo(N), next_pow2s = getCeilPowerOfTwo(s), new_rows = s*(1-c), old_rows = s - new_rows, max_fit_offset = 0;

    float min11 = 0, max_fit = 0, sum = 0;
    individual_fit<<<s,next_pow2N,next_pow2N*sizeof(float)>>>(new_pop_ptr,city_dis_ptr, new_fit_ptr, N, s);

    sum = thrust::reduce(thrust::device, new_fit.begin(),new_fit.end());

    time_t t;
    srand((unsigned int)time(&t));

    for(int time = 0;time<times;time++){
        old_pop = new_pop;
        old_fit = new_fit;
        min11 = *thrust::min_element(thrust::device, new_fit.begin(), new_fit.end());
        thrust::generate(thrust::host, h_prob_a.begin(), h_prob_a.end(), rand_01);
        thrust::generate(thrust::host, h_prob_b.begin(), h_prob_b.end(), rand_01);
        thrust::generate(thrust::host, h_cross_pos.begin(), h_cross_pos.end(), rand_N);
        thrust::generate(thrust::host, h_mutation_pos.begin(), h_mutation_pos.end(), rand_N);
        d_prob_a = h_prob_a;
        d_prob_b = h_prob_b;
        d_cross_pos = h_cross_pos;
        d_mutation_pos = h_mutation_pos;

        if(time%100==0) std::cout<<"iter: "<<time<<", max fit:"<<max_fit<<std::endl;
        thrust::transform(thrust::device, new_fit.begin(), new_fit.begin()+int(s/2), new_fit.begin()+int(s/2), pc1.begin(), [=]__device__(float fit1, float fit2)->float{
        float bj = fit1<fit2?fit1:fit2;
        if(bj<=sum/s)   return pc*(bj-min11)/(sum/s-min11);
        else            return pc;
        });
        thrust::transform(thrust::device, new_fit.begin(), new_fit.end(), pm1.begin(), [=]__device__(float fit){
        if(fit<=sum/s)  return pm*(fit-min11)/(sum/s-min11);
        else            return pm;
        });
        // CrossVariation
        crossover<<<s,N,5*N*sizeof(int)>>>(new_pop_ptr, pc1_ptr, cross_pos_ptr, prob_a_ptr, N, s);

        // Mutation
        mutation<<<s,N,N*sizeof(int)>>>(new_pop_ptr, pm1_ptr, mutation_pos_ptr, prob_b_ptr, N,s);

        // GroupFit
        individual_fit<<<s,next_pow2N,next_pow2N*sizeof(float)>>>(new_pop_ptr,city_dis_ptr, new_fit_ptr, N, s);

        // ChooseParents
        old_ind = ind;
        new_ind = ind;
        thrust::sort_by_key(thrust::device, old_fit.begin(), old_fit.end(), old_ind.begin(), thrust::greater<float>());
        thrust::sort_by_key(thrust::device, new_fit.begin(), new_fit.end(), new_ind.begin(), thrust::greater<float>());
        choose_parents<<<s,N>>>(old_pop_ptr, new_pop_ptr, old_ind_ptr, new_ind_ptr, old_fit_ptr, new_fit_ptr, old_rows, new_rows, N, s);

        thrust::device_vector<float>::iterator max_ptr = thrust::max_element(thrust::device, new_fit.begin(), new_fit.end());
        float max_val = *max_ptr;

        if(max_val>max_fit){
            std::cout<<"save new pop_min"<<std::endl;
            max_fit_offset = max_ptr - new_fit.begin();
            max_fit = max_val;
        }
    }
    thrust::copy(thrust::device, new_pop.begin()+N*max_fit_offset, new_pop.begin()+N*(max_fit_offset+1), pop_min.begin());
    thrust::host_vector<int> fittest_pop = pop_min;
    return std::move(fittest_pop);
}

void TSP_GA::SaveResult(std::string filepath, thrust::host_vector<int>& res){
    std::ofstream fp(filepath, std::ios::out);
    if(!fp.is_open()){
        std::cout<<"Error: saving result fail"<<std::endl;
        std::exit(1);
    }
    for(int i = 0;i<res.size();i++){
        // std::cout<<res[i]<<std::endl;
        fp<<res[i]<<" ";
    }
    fp.close();
    std::cout<<"solution saved!"<<std::endl;
}   